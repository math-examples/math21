/* Copyright 2015 The math21 Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include "matrix_cuda.h"

void math21_matrix_multiply_k1AB_add_k2C_similar_cuda(int ta, int tb, int nr_C, int nc_C, int n_common, float k1,
                                                      const float *A, int stride_a,
                                                      const float *B, int stride_b,
                                                      float k2,
                                                      float *C, int stride_c) {
    hipblasHandle_t handle = math21_cuda_blas_handle();
    hipblasStatus_t status = hipblasSgemm(handle, (tb ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                     (ta ? HIPBLAS_OP_T : HIPBLAS_OP_N), nc_C, nr_C, n_common, &k1, B, stride_b, A,
                                     stride_a, &k2, C, stride_c);
    math21_cuda_cublas_check_error(status);
}
